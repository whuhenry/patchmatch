#include "hip/hip_runtime.h"
#include "patchmatch_gpu.h"

#include <hip/hip_runtime.h>
#include <>

void init(Image* im, PatchMatchConfig cfg) {
    int pixel_count = im->rows_ * im->cols_;
    hipMalloc(&(im->d_image_), pixel_count * 3 * sizeof(float));
    hipMemcpy(im->d_image_, im->image_, pixel_count * 3 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&(im->d_plane_), pixel_count * 3 * sizeof(float));
    hipMalloc(&(im->d_normal_), pixel_count * 3 * sizeof(float));
    hipMalloc(&(im->d_grad_), pixel_count * 3 * sizeof(float));
    hipMalloc(&(im->d_cost_), pixel_count * sizeof(float));

    dim3 grid_size, blockdim(BLOCK_DIM_SIZE, BLOCK_DIM_SIZE);
    grid_size.x = (cfg.cols + blockdim.x - 1) / blockdim.x;
    grid_size.y = (cfg.cols + blockdim.y - 1) / blockdim.y;
    hiprandState *devStates;
    hipMalloc((void **)&devStates, blockdim.x * blockdim.y * sizeof(hiprandState));

    initNormalAndPlane<<<grid_size, blockdim>>>(im->d_normal_, im->d_plane_, devStates, cfg);
    
}

__global__ void initNormalAndPlane(float* normal, float* plane, hiprandState* globalState, PatchMatchConfig cfg) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * cfg.cols + x;

    hiprandState local_state;
    local_state = globalState[threadIdx.x];
    
    float disp = hiprand_uniform(&local_state) * cfg.max_disp;
    normal[offset]     = (hiprand_uniform(&local_state) - 0.5f) * 2;
    normal[offset + 1] = (hiprand_uniform(&local_state) - 0.5f) * 2;
    normal[offset + 2] = hiprand_uniform(&local_state);
    norm(normal + offset);

    normal_to_plane(x, y, disp, normal + offset, plane + offset);
}