#include "hip/hip_runtime.h"
#include "patchmatch_gpu.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"

void init(Image* im, PatchMatchConfig cfg) {
    int pixel_count = im->rows_ * im->cols_;
    checkCudaErrors(hipMalloc(&(im->d_image_), pixel_count * 3 * sizeof(float)));
    checkCudaErrors(hipMemcpy(im->d_image_, im->image_, pixel_count * 3 * sizeof(float), hipMemcpyHostToDevice));

    im->plane_ = new float[pixel_count * 3];
    for (int i = 0; i < pixel_count * 3; ++i) {
        im->plane_[i] = 0.0f;
    }

    checkCudaErrors(hipMalloc(&(im->d_plane_), pixel_count * 3 * sizeof(float)));
    checkCudaErrors(hipMemcpy(im->d_plane_, im->plane_, pixel_count * 3 * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&(im->d_normal_), pixel_count * 3 * sizeof(float)));
    checkCudaErrors(hipMalloc(&(im->d_grad_), pixel_count * 3 * sizeof(float)));
    checkCudaErrors(hipMalloc(&(im->d_cost_), pixel_count * sizeof(float)));

    
    hiprandState *devStates;
    hipError_t err = hipMalloc(&devStates, 64 * 64 * sizeof(hiprandState));
    setup_kernel<<<64, 64>>>(devStates);
 
    int pixel_count_per_thread = (cfg.rows * cfg.cols + 64 * 64 - 1) / (64 * 64);
    initNormalAndPlane<<<64, 64>>>(im->d_normal_, im->d_plane_, im->d_cost_, devStates, cfg, pixel_count_per_thread);
    checkCudaErrors(hipFree(devStates));

//    checkCudaErrors(hipMemcpy(im->plane_, im->d_plane_, pixel_count * 3 * sizeof(float), hipMemcpyDeviceToHost));
//    cv::Mat disp(cfg.rows, cfg.cols, CV_8U);
//    int offset = 0;
//    for (int i = 0; i < cfg.rows; ++i) {
//        for (int j = 0; j < cfg.cols; ++j) {
//            disp.at<uint8_t>(i, j) = (uint8_t)(j * im->plane_[offset] + i * im->plane_[offset + 1] + im->plane_[offset + 2] / cfg.max_disp * 255.0f);
//            offset += 3;
//        }
//    }
//    cv::imshow("disp", disp);
//    cv::waitKey(0);
}

void solve(Image * im_left, Image * im_right, PatchMatchConfig cfg)
{
    dim3 grid_size, blockdim(BLOCK_DIM_SIZE, BLOCK_DIM_SIZE);
    grid_size.x = (cfg.cols + blockdim.x - 1) / blockdim.x;
    grid_size.y = (cfg.rows + blockdim.y - 1) / blockdim.y;
    cuImage cuim_left, cuim_right;
    cpy_host_image_to_cuimage(im_left, &cuim_left);
    cpy_host_image_to_cuimage(im_right, &cuim_right);

    checkCudaErrors(hipMalloc(&cfg.d_neighbor_lists, cfg.neighbor_lists_len * sizeof(int2)));
    checkCudaErrors(hipMemcpy(cfg.d_neighbor_lists, cfg.h_neighbor_lists, 
                               cfg.neighbor_lists_len * sizeof(int2), hipMemcpyHostToDevice));

    for (int iter = 0; iter < 1/*cfg.iter_count*/; ++iter) {
        //left to right red
        spatialPropagation<<<grid_size, blockdim>>>(cuim_left, cuim_right, cfg, 0, 1);
        ////left to right black
        spatialPropagation <<<grid_size, blockdim>>>(cuim_left, cuim_right, cfg, 1, 1);
        ////right to left black
        //spatialPropagation <<<grid_size, blockdim>>>(im_right, im_left, cfg, 0, -1);
        ////right to left black
        //spatialPropagation <<<grid_size, blockdim>>>(im_right, im_left, cfg, 1, -1);
    }
    //hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(im_left->plane_, im_left->d_plane_, 
                               cfg.rows * cfg.cols * 3 * sizeof(float), hipMemcpyDeviceToHost));
//    cv::Mat disp(cfg.rows, cfg.cols, CV_8U);
//    int offset = 0;
//    for (int i = 0; i < cfg.rows; ++i) {
//        for (int j = 0; j < cfg.cols; ++j) {
//            disp.at<uint8_t>(i, j) = (uint8_t)(j * im_left->plane_[offset] + i * im_left->plane_[offset + 1]
//                                               + im_left->plane_[offset + 2] / cfg.max_disp * 255.0f);
//            offset += 3;
//        }
//    }
//    cv::imshow("disp", disp);
//    cv::waitKey(0);
}

void cpy_host_image_to_cuimage(Image* host_im, cuImage* cu_im) {
    cu_im->d_image = host_im->d_image_;
    cu_im->d_plane = host_im->d_plane_;
    cu_im->d_normal = host_im->d_normal_;
    cu_im->d_grad = host_im->d_grad_;
    cu_im->d_cost = host_im->d_cost_;
}

__global__ void setup_kernel(hiprandState* state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(0, id, 0, &state[id]);
}

__global__ void initNormalAndPlane(float* normal, float* plane, float* cost, hiprandState* globalState,
                                   PatchMatchConfig cfg, int pixel_per_thread) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_offset = tid * pixel_per_thread;
    int offset = pixel_offset * 3;

    hiprandState local_state = globalState[tid];
    int y = pixel_offset / cfg.cols;
    int x = pixel_offset % cfg.cols;
    if (y >= cfg.rows) {
        return;
    }

    for (int i = 0; i < pixel_per_thread; ++i) {        
        float disp = hiprand_uniform(&local_state) * cfg.max_disp;
        normal[offset] = (hiprand_uniform(&local_state) - 0.5f) * 2;
        normal[offset + 1] = (hiprand_uniform(&local_state) - 0.5f) * 2;
        normal[offset + 2] = hiprand_uniform(&local_state);
        norm(normal + offset);

        normal_to_plane(x, y, disp, normal + offset, plane + offset);
        cost[pixel_offset] = cfg.max_cost_single * (cfg.window_radius * 2 + 1) * (cfg.window_radius * 2 + 1);
        offset += 3;
        ++x;
        if (x >= cfg.cols) {
            x = 0;
            ++y;
            if(y >= cfg.rows) {
                return;
            }
        }
    }
}

__global__ void spatialPropagation(cuImage im_base, cuImage im_ref, PatchMatchConfig cfg, int red_or_black, int direction) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= cfg.cols || y >= cfg.rows) {
        return;
    }
    int offset = y * cfg.cols + x;

    //chess board update: detail see paper "Massively Parallel Multiview Stereopsis by Surface Normal Diffusion"
    if((y % 2 + x % 2) % 2 != red_or_black) {
        return;
    }

    float* center_plane = &im_base.d_plane[offset * 3];
    float* max_cost = &im_base.d_cost[offset];
    for (int i = 0; i < cfg.neighbor_lists_len; ++i) {
        int ny = y + cfg.d_neighbor_lists[i].y;
        int nx = x + cfg.d_neighbor_lists[i].x;
        if (ny <= 0 || ny >= cfg.rows || nx <= 0 || ny >= cfg.cols) {
            continue;
        }
        float* comp_plane = &im_base.d_plane[(ny * cfg.cols + nx) * 3];
        float cost = compute_cost_cu(&im_base, &im_ref, x, y, comp_plane, direction, &cfg);
        
        if (cost < max_cost[0]) {
            max_cost[0] = cost;
            cpy_vec3(center_plane, comp_plane);
        }
    }

//    if (x > 400 && y >300) {
//        float weight = im_base.d_plane[(y * cfg.cols + x) * 3] - im_base.d_plane[(y * cfg.cols + x) * 3];
//        //printf("%d, %d, %f\n", x, y, im_base.d_plane[(y * cfg.cols + x) * 3] - im_base.d_plane[(y * cfg.cols + x) * 3]);
//        im_base.d_plane[offset * 3] = 0.0f;
//        im_base.d_plane[offset * 3 + 1] = 0.0f;
//        im_base.d_plane[offset * 3 + 2] = 0.0f;
//    }
    
}

__device__ float compute_cost_cu(cuImage* im_base, cuImage* im_ref, int x, int y,
                                 float* plane_used, int direction, PatchMatchConfig *cfg) {
    if (y - cfg->window_radius < 0 || y + cfg->window_radius > cfg->rows - 1
        || x - cfg->window_radius < 0 || x + cfg->window_radius > cfg->cols - 1) {
        return cfg->max_cost_single * (cfg->window_radius * 2 + 1) * (cfg->window_radius * 2 + 1);
    } else {
        int x_st = x - cfg->window_radius;
        int x_ed = x + cfg->window_radius;
        int y_st = y - cfg->window_radius;
        int y_ed = y + cfg->window_radius;
        float sum_cost = 0.0f;
        int center_offset = (y * cfg->cols + x) * 3;
        float density_ref_single[3], grad_ref_single[3];
        int cw_offset;
        float weight, disp, cor_x;
        for (int cw_y = y_st; cw_y <= y_ed; ++cw_y) {
            for (int cw_x = x_st; cw_x <= x_ed; ++cw_x) {
                cw_offset = (cw_y * cfg->cols + cw_x) * 3;
                //weight = im_base->d_plane[center_offset] - im_base->d_plane[center_offset];
                //float weight = abs(im_base.d_plane[center_offset] - im_base.d_plane[cw_offset]);
                weight = exp(-l1_distance(&im_base->d_image[center_offset], &im_base->d_image[cw_offset]) / cfg->gamma);
                disp = plane_to_disp(cw_x, cw_y, plane_used);
                cor_x = cw_x - disp * direction;
                if (cor_x < 0 || cor_x > cfg->cols - 1) {
                    sum_cost += weight * cfg->max_cost_single;
                } else {
                    get_value_bilinear(cor_x, cw_y, cfg, im_ref->d_image, density_ref_single);
                    get_value_bilinear(cor_x, cw_y, cfg, im_ref->d_grad, grad_ref_single);
                    sum_cost += weight * (
                        (1 - cfg->alpha)
                        * MIN(l1_distance(im_base->d_image + cw_offset, density_ref_single), cfg->density_diff_max)
                        + cfg->alpha
                        * MIN(l1_distance(im_base->d_grad + cw_offset, grad_ref_single), cfg->grad_diff_max));
                }
            }
        }
        return sum_cost;
    }
}
